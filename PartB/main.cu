#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <ctime>
#include <chrono>
#include <fstream>
#include <assert.h>
using namespace std;

#include "gpu_thread.h"

// Used to cross-check answer. DO NOT MODIFY!
void reference(int N, int *matA, int *matB, int *output)
{
  // enforce N to be power of 2 and greater than 2
  assert( N>=4 and N == ( N &~ (N-1)));
  for(int rowA = 0; rowA < N; rowA +=2) {
    for(int colB = 0; colB < N; colB += 2){
      int sum = 0;
      for(int iter = 0; iter < N; iter++) 
      {
        sum += matA[rowA * N + iter] * matB[iter * N + colB];
        sum += matA[(rowA+1) * N + iter] * matB[iter * N + colB];
        sum += matA[rowA * N + iter] * matB[iter * N + (colB+1)];
        sum += matA[(rowA+1) * N + iter] * matB[iter * N + (colB+1)];
      }

      // compute output indices
      int rowC = rowA>>1;
      int colC = colB>>1;
      int indexC = rowC * (N>>1) + colC;
      output[indexC] = sum;
    }
  }
}

int main(int argc, char *argv[])
{
    // Input size of square matrices// Input size of square matrices
    int N;
    string file_name; 
    if (argc < 2) 
        file_name = "data/input_128.in"; 
    else 
        file_name = argv[1]; 
    ifstream input_file; 
    input_file.open(file_name); 
    input_file >> N;
    cout << "Input matrix of size " << N << "\n";
    
    // Input matrix A
    int *matA = new int[N * N];
    for(int i = 0; i < N; ++i)
        for(int j = 0; j < N; ++j)
            input_file >> matA[i * N + j];

    // Input matrix B
    int *matB = new int[N * N];
    for(int i = 0; i < N; ++i)
        for(int j = 0; j < N; ++j)
            input_file >> matB[i * N + j];
    
    // Execute reference program
    int *output_reference = new int[N*(N>>1)];
    reference(N, matA, matB, output_reference);
    
    // Execute gpu version
    int *output_gpu = new int[N*(N>>1)];
    gpuThread(N, matA, matB, output_gpu);
    
    for(int i = 0; i < N*(N>>1); ++i)
        if(output_gpu[i] != output_reference[i]) {
            cout << "Mismatch at " << i << "\n";
            cout << "GPU output: " << output_gpu[i] << ", required output: " << output_reference[i] << "\n";
            exit(0);
        }
    input_file.close(); 
}
